#include "hip/hip_runtime.h"
#include <SLutils/multifrequency.hpp>

#include <SLutils/fringe_analysis.hpp> // NStepPhaseShifting

#include <opencv2/core/cuda.hpp>


namespace sl {

__global__ void equivalentPhase(const cv::cuda::PtrStepSz<double> phase1,
                                const cv::cuda::PtrStep<double> phase2,
                                cv::cuda::PtrStep<double> eqPhase) {
    int j = blockIdx.x*blockDim.x + threadIdx.x;
    int i = blockIdx.y*blockDim.y + threadIdx.y;
    if (i >= phase1.rows || j >= phase1.cols) return;
    
    constexpr double twoPI = 2*CV_PI;
    
    // Phase difference
    double diff = phase1(i,j) - phase2(i,j);
    
    // Estimate modulus
    double mod = fmodf(diff, twoPI);
    if (mod < 0) mod += twoPI;

    eqPhase(i,j) = mod;
}

__global__ void backwardUnwrap(const cv::cuda::PtrStepSz<double> phase1,
                               cv::cuda::PtrStep<double> phase2, double T1, double T2) {
    int j = blockIdx.x*blockDim.x + threadIdx.x;
    int i = blockIdx.y*blockDim.y + threadIdx.y;
    if (i >= phase1.rows || j >= phase1.cols) return;
    
    constexpr double twoPI = 2*CV_PI;
    
    double phi2 = phase2(i,j);
    
    // Estimate phase order
    double k = (T1/T2*phase1(i,j) - phi2)/twoPI;
    
    // Unwrap phase value
    phase2(i,j) = phi2 + twoPI*round(k);
}

__global__ void removeSpikyNoise(cv::cuda::PtrStepSz<double> Phi) {
    int j = blockIdx.x*blockDim.x + threadIdx.x;
    int i = blockIdx.y*blockDim.y + threadIdx.y;

    constexpr int ksize{5};
    int mid = ksize/2;
    if (i < mid || i > Phi.rows-1-mid || j < mid || j > Phi.cols-1-mid) return;

    // -------------------------- Median filter
    double values[ksize*ksize];
    for (int row = 0; row < ksize; row++) {
        for (int col = 0; col < ksize; col++) {
            int m = row*ksize + col;
            values[m] = Phi(i+row-mid, j+col-mid);

            // Sorting the elements (Insertion Sort)
            if (m != 0) {
                double v = values[m];
                int n = m - 1;
                while (n >= 0 && values[n] > v) {
                    values[n+1] = values[n];
                    n--;
                }
                values[n+1] = v;
            }
        }
    }

    // Get the median phase value at (i,j)
    double Phim = values[ksize*ksize/2];

    // -------------------------- Remove spiky points
    // Determine order of 2*pi to add to remove spiky points
    Phi(i,j) -= 2*CV_PI*round( (Phi(i,j) - Phim)/2/CV_PI );
}

void threeFreqPhaseUnwrap(const std::vector<std::string>& impaths, cv::OutputArray _Phi,
                          const cv::Vec3i& p, const cv::Vec3i& N) {
    if (impaths.size() != (N[0]+N[1]+N[2]))
        throw std::runtime_error("threeFreqPhaseUnwrap: number of image paths and number of patterns N must match.");
    
    // Get input fringe periods
    double T1 = p[0], T2 = p[1], T3 = p[2];
    // Estimate equivalent intermidate periods
    double T12 = T1*T2/std::abs(T1-T2);
    double T23 = T2*T3/std::abs(T2-T3);
    double T123 = T12*T3/std::abs(T12-T3);
    
    // ------------- Estimating wrapped phase map for each frequency
    cv::cuda::GpuMat phi1, phi2, phi3;
    NStepPhaseShifting({impaths.begin(), impaths.begin()+N[0]}, phi1, N[0]);
    NStepPhaseShifting({impaths.begin()+N[0], impaths.begin()+N[0]+N[1]}, phi2, N[1]);
    NStepPhaseShifting({impaths.end()-N[2], impaths.end()}, phi3, N[2]);
    

    // ------------- Estimate equivalent phase maps
    dim3 block(16, 16);
    dim3 grid((phi1.cols + block.x - 1)/block.x, (phi1.rows + block.y - 1)/block.y);
    
    cv::cuda::GpuMat phi12(phi1.size(), phi1.type());
    equivalentPhase<<<grid, block>>>(phi1, phi2, phi12);
    
    cv::cuda::GpuMat phi23(phi1.size(), phi1.type());
    equivalentPhase<<<grid, block>>>(phi2, phi3, phi23);
    
    cv::cuda::GpuMat Phi123(phi1.size(), phi1.type());
    equivalentPhase<<<grid, block>>>(phi12, phi3, Phi123); // Phi123 is a wide phase without discontinuities
    
    
    // ------------- Remove spiky noise in the equivalent phase of wider pitch
    removeSpikyNoise<<<grid, block>>>(Phi123);

    
    // ------------- Backward phase unwrapping
    backwardUnwrap<<<grid, block>>>(Phi123, phi23, T123, T23); // Estimate unwrapped version of phi23
    backwardUnwrap<<<grid, block>>>(phi23, phi12, T23, T12); // Estimate unwrapped version of phi12
    backwardUnwrap<<<grid, block>>>(phi12, phi3, T12, T3); // Estimate unwrapped version of phi3
    backwardUnwrap<<<grid, block>>>(phi3, phi2, T3, T2); // Estimate unwrapped version of phi2
    backwardUnwrap<<<grid, block>>>(phi2, phi1, T2, T1); // Estimate unwrapped version of phi1
    
    phi1.copyTo(_Phi);
}

void twoFreqPhaseUnwrap(const std::vector<std::string>& impaths, cv::OutputArray _Phi,
                        const cv::Vec3i& p, const cv::Vec3i& N) {
    if (impaths.size() != (N[0]+N[1]))
        throw std::runtime_error("twoFreqPhaseUnwrap: number of image paths and number of patterns N must match.");
    
    // Get input fringe periods
    double T1 = p[0], T2 = p[1];
    // Estimate equivalent period
    double T12 = T1*T2/std::abs(T1-T2);
    
    // Estimating wrapped phase map for each frequency
    cv::cuda::GpuMat phi1, phi2;
    NStepPhaseShifting({impaths.begin(), impaths.begin()+N[0]}, phi1, N[0]);
    NStepPhaseShifting({impaths.begin()+N[0], impaths.end()}, phi2, N[1]);
    

    // Estimate equivalent phase map
    dim3 block(16, 16);
    dim3 grid((phi1.cols + block.x - 1)/block.x, (phi1.rows + block.y - 1)/block.y);
    
    cv::cuda::GpuMat Phi12(phi1.size(), phi1.type());
    equivalentPhase<<<grid, block>>>(phi1, phi2, Phi12); // Phi12 is a phase map without discontinuities
    
    
    // Remove spiky noise in the equivalent phase of wider pitch
    removeSpikyNoise<<<grid, block>>>(Phi12);
    
    // Backward phase unwrapping
    backwardUnwrap<<<grid, block>>>(Phi12, phi2, T12, T2); // Estimate unwrapped version of phi2
    backwardUnwrap<<<grid, block>>>(phi2, phi1, T2, T1); // Estimate unwrapped version of phi1
    
    phi1.copyTo(_Phi);
}

} // namespace sl
