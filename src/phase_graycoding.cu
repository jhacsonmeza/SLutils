#include "hip/hip_runtime.h"
#include <SLutils/phase_graycoding.hpp>

#include <SLutils/fringe_analysis.hpp> // NStepPhaseShifting
#include <SLutils/graycoding.hpp> // decimalMap

#include <opencv2/core/cuda.hpp>


namespace sl {

__global__ void unwrapWithPhaseOrder(const cv::cuda::PtrStepSz<double> phi, const cv::cuda::PtrStepi k,
                                     cv::cuda::PtrStep<double> Phi, double shift) {
    int j = blockIdx.x*blockDim.x + threadIdx.x;
    int i = blockIdx.y*blockDim.y + threadIdx.y;
    if (i >= phi.rows || j >= phi.cols) return;

    // Shift and rewrap wrapped phase values
    double phi_ij = phi(i,j) + shift;
    double phi_shifted = atan2(sin(phi_ij), cos(phi_ij));

    // Estimate unwrapped phase map with k order map: Phi = phi + 2*pi*k
    double Phi_shifted = phi_shifted + 2*CV_PI*k(i,j);

    // Shift phase back to the original values
    Phi(i,j) = Phi_shifted - shift;
}

__global__ void removeSpikyNoise(cv::cuda::PtrStepSz<double> Phi) {
    int j = blockIdx.x*blockDim.x + threadIdx.x;
    int i = blockIdx.y*blockDim.y + threadIdx.y;

    constexpr int ksize{5};
    int mid = ksize/2;
    if (i < mid || i > Phi.rows-1-mid || j < mid || j > Phi.cols-1-mid) return;

    // -------------------------- Median filter
    double values[ksize*ksize];
    for (int row = 0; row < ksize; row++)
        for (int col = 0; col < ksize; col++)
        {
            int m = row*ksize + col;
            values[m] = Phi(i+row-mid, j+col-mid);

            // Sorting the elements (Insertion Sort)
            if (m != 0)
            {
                double v = values[m];
                int n = m - 1;
                while (n >= 0 && values[n] > v)
                {
                    values[n+1] = values[n];
                    n--;
                }
                values[n+1] = v;
            }
        }

    // Get the median phase value at (i,j)
    double Phim = values[ksize*ksize/2];

    // -------------------------- Remove spiky points
    // Determine order of 2*pi to add to remove spiky points
    Phi(i,j) -= 2*CV_PI*round( (Phi(i,j) - Phim)/2/CV_PI );
}

void phaseGraycodingUnwrap(const std::vector<std::string>& impaths_ps,
                           const std::vector<std::string>& impaths_gc,
                           cv::OutputArray _Phi, int p, int N) {
    // Estimate wrapped phase map
    cv::cuda::GpuMat phi; // double mat
    NStepPhaseShifting(impaths_ps, phi, N);
    
    // Estimate decimal map (phase order) with the gray patterns
    cv::cuda::GpuMat k;
    decimalMap(impaths_gc, k);


    // --- Phase unwrapping using the phase order map k
    dim3 block(16, 16);
    dim3 grid((phi.cols + block.x - 1)/block.x, (phi.rows + block.y - 1)/block.y);
    double shift = -CV_PI + CV_PI/p;
    // Get output array
    _Phi.create(phi.size(), phi.type());
    cv::cuda::GpuMat Phi = _Phi.getGpuMat();
    // Launch kernel
    unwrapWithPhaseOrder<<<grid, block>>>(phi, k, Phi, shift);


    // --- Remove spiky noise using median filter
    removeSpikyNoise<<<grid, block>>>(Phi);
}

} // namespace sl
