#include "hip/hip_runtime.h"
#include <SLutils/graycoding.hpp>

#include <opencv2/core/cuda.hpp>
#include <stdexcept> // std::runtime_error


namespace sl {

__global__ void initDecimalArray(const cv::cuda::PtrStepSzb gray, cv::cuda::PtrStepi decimal, int n_bits) {
    int j = blockIdx.x*blockDim.x + threadIdx.x;
    int i = blockIdx.y*blockDim.y + threadIdx.y;
    if (i >= gray.rows || j >= gray.cols) return;

    decimal(i,j) = gray(i,j) ? 1 << (n_bits - 1) : 0;
}

__global__ void gray2dec_array(const cv::cuda::PtrStepSzb gray, cv::cuda::PtrStepb bin,
                               cv::cuda::PtrStepi decimal, int n_bits, int pos) {
    int j = blockIdx.x*blockDim.x + threadIdx.x;
    int i = blockIdx.y*blockDim.y + threadIdx.y;
    if (i >= gray.rows || j >= gray.cols) return;

    // Convert current gray code bit to binary bit using xor between 
    // the previous binary bit and the current gray bit
    // see: https://www.geeksforgeeks.org/gray-to-binary-and-binary-to-gray-conversion/
    bin(i,j) ^= gray(i,j);
    // if binary bit is 1 then add 2^(bit_pos) to the decimal array
    if (bin(i,j)) decimal(i,j) += 1 << (n_bits - pos - 1);
}


void decimalMap(const std::vector<std::string>& impaths, cv::OutputArray _dec) {
    if (impaths.size() > 1 and impaths.size() % 2 != 0)
        throw std::runtime_error("decimalMap requires an even set of images");

    cv::cuda::Stream stream0;
    
    // Total number of graycode bits (pairs of captured graycode patterns)
    int n = impaths.size()/2;
    

    /* -----------------------------------------------------------------------
    Initialize decimal array (phase order map) using the first pair of
    graycode images. Also the binary map, which is equal to the graycode map
    because the Most Significant Bit (MSB) of the binary code = MSB gray code
    ----------------------------------------------------------------------- */
    cv::Mat im1_h = cv::imread(impaths[0], 0);
    cv::cuda::GpuMat im1;
    im1.upload(im1_h, stream0);
    
    cv::Mat im2_h = cv::imread(impaths[1], 0);
    cv::cuda::GpuMat im2;
    im2.upload(im2_h, stream0);

    // Allocate output decimal array which is obtained from graycode words
    _dec.create(im1_h.size(), CV_32S);
    cv::cuda::GpuMat dec = _dec.getGpuMat();
    
    // Allocate binary array
    cv::cuda::GpuMat bin(im1_h.size(), CV_8U);

    // Launching initDecimalAndBinary to initialize the values of dec and bin
    dim3 block(16, 16);
    dim3 grid((dec.cols + block.x - 1)/block.x, (dec.rows + block.y - 1)/block.y);
    initDecimalAndBinary<<<grid, block>>>(im1, im2, dec, bin, n);


    /* -----------------------------------------------------------------------
    Adding the rest of graycode patterns to estimate the final phase order
    -------------------------------------------------------------------------- */
    for (int i = 1; i < n; i++) {
        // Read graycoding pattern and its inverted counterpart
        cv::Mat im1_h = cv::imread(impaths[2*i], 0);
        cv::cuda::GpuMat im1;
        im1.upload(im1_h, stream0);
        
        cv::Mat im2_h = cv::imread(impaths[2*i+1], 0);
        cv::cuda::GpuMat im2;
        im2.upload(im2_h, stream0);

        dec_array<<<grid, block>>>(im1, im2, bin, dec, n, i);
    }
}

void graycodeword(const std::vector<std::string>& impaths, cv::OutputArray _code_word) {
    if (impaths.size() > 1 and impaths.size() % 2 != 0)
        throw std::runtime_error("graycodeword requires an even set of images");

    cv::cuda::Stream stream0;
    
    // Total number of graycode bits (pairs of captured graycode patterns)
    int n = impaths.size()/2;

    // Read first image to obtain the output array size
    cv::Size sz = cv::imread(impaths[0], 0).size();

    // Get output vector of arrays
    std::vector<cv::cuda::GpuMat>& gray_images = _code_word.getGpuMatVecRef();

    for (int k = 0; k < n; k++) {
        // Read graycoding pattern and its inverted counterpart
        cv::Mat im1 = cv::imread(impaths[2*k], 0);
        cv::Mat im2 = cv::imread(impaths[2*k+1], 0);
        // Generate a single gray map
        cv::Mat gray_h = (im1 > im2)/255;

        // Convert to GPU with continuous memory block of byte data
        cv::cuda::GpuMat gray;
        gray.upload(gray_h, stream0);
        gray_images.push_back(gray);
    }
}

void gray2dec(cv::InputArray _code_word, cv::OutputArray _dec) {
    // Obtain input vector of GpuMats
    std::vector<cv::cuda::GpuMat> code_word;
    _code_word.getGpuMatVector(code_word);
    if (code_word.size() < 2)
        throw std::runtime_error("gray2dec needs at least more than 1 graycode map");

    // Number of graycoding arrays, rows, and columns of the images
    int n = code_word.size(), h = code_word[0].rows, w = code_word[0].cols;


    // Output array that store graycode words converted to decimal
    _dec.create(h, w, CV_32S);
    cv::cuda::GpuMat dec = _dec.getGpuMat();

    // Launching initDecimalArray to initialize the values of dec
    dim3 block(16, 16);
    dim3 grid((w + block.x - 1)/block.x, (h + block.y - 1)/block.y);
    initDecimalArray<<<grid, block>>>(code_word[0], dec, n);

    // Initializing the binary map. 
    // Where the Most Significant Bit (MSB) of the binary code = MSB gray code
    cv::cuda::GpuMat bin = code_word[0].clone();
    // Convert from gray code to decimal
    for (int i = 1; i < n; i++)
        gray2dec_array<<<grid, block>>>(code_word[i], bin, dec, n, i);
}

} // namespace sl
