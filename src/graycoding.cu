#include "hip/hip_runtime.h"
#include <phase_unwrap/graycoding.hpp>

#include <opencv2/cudaarithm.hpp>
#include <stdexcept> // std::runtime_error


namespace sl {

__global__ void initDecimalArray(const cv::cuda::PtrStepSzb gray, cv::cuda::PtrStepi decimal, int n_bits) {
    int j = blockIdx.x*blockDim.x + threadIdx.x;
    int i = blockIdx.y*blockDim.y + threadIdx.y;
    if (i >= gray.rows || j >= gray.cols) return;

    decimal(i,j) = gray(i,j) ? 1 << (n_bits - 1) : 0;
}

__global__ void gray2dec_array(const cv::cuda::PtrStepSzb gray, cv::cuda::PtrStepb bin,
                               cv::cuda::PtrStepi decimal, int n_bits, int pos) {
    int j = blockIdx.x*blockDim.x + threadIdx.x;
    int i = blockIdx.y*blockDim.y + threadIdx.y;
    if (i >= gray.rows || j >= gray.cols) return;

    // Convert current gray code bit to binary bit using xor between 
    // the previous binary bit and the current gray bit
    // see: https://www.geeksforgeeks.org/gray-to-binary-and-binary-to-gray-conversion/
    bin(i,j) ^= gray(i,j);
    // if binary bit is 1 then add 2^(bit_pos) to the decimal array
    if (bin(i,j)) decimal(i,j) += 1 << (n_bits - pos - 1);
}


void decimalMap(const std::vector<std::string>& imlist, cv::OutputArray _dec) {
    if (imlist.size() % 2 != 0)
        throw std::runtime_error("decimalMap requires an even set of images\n");

    cv::cuda::Stream stream0;
    
    // Total number of graycode bits (pairs of captured graycode patterns)
    int n = imlist.size()/2;
    

    /* -----------------------------------------------------------------------
    Initialize decimal array (phase order map) 
    using the first pair of graycode images
    ----------------------------------------------------------------------- */
    cv::Mat im1 = cv::imread(imlist[0], 0);
    cv::Mat im2 = cv::imread(imlist[1], 0);
    cv::Mat gray_h = (im1 > im2)/255;
    // Convert to GPU
    cv::cuda::GpuMat gray;
    gray.upload(gray_h, stream0);

    // Create output array that stores graycode words converted to decimal
    _dec.create(im1.size(), CV_32S);
    cv::cuda::GpuMat dec = _dec.getGpuMat();

    // Launching initDecimalArray to initialize the values of dec
    dim3 block(16, 16);
    dim3 grid((dec.cols + block.x - 1)/block.x, (dec.rows + block.y - 1)/block.y);
    initDecimalArray<<<grid, block>>>(gray, dec, n);

    
    /* -----------------------------------------------------------------------
    Initializing the binary map, which is equal to the graycode map
    because the Most Significant Bit (MSB) of the binary code = MSB gray code
    -------------------------------------------------------------------------- */
    cv::cuda::GpuMat bin = gray.clone();


    /* -----------------------------------------------------------------------
    Adding the rest of graycode patterns to estimate the final phase order
    -------------------------------------------------------------------------- */
    for (int i = 1; i < n; i++) {
        // Read graycoding pattern and its inverted counterpart
        cv::Mat im1 = cv::imread(imlist[2*i], 0);
        cv::Mat im2 = cv::imread(imlist[2*i+1], 0);
        // Generate a single gray map
        cv::Mat gray_h = (im1 > im2)/255;
        // Conver to GPU with continuous memory block of byte data
        cv::cuda::GpuMat gray;
        gray.upload(gray_h, stream0);

        gray2dec_array<<<grid, block>>>(gray, bin, dec, n, i);
    }
}

void graycodeword(const std::vector<std::string>& imlist, cv::OutputArray _code_word) {
    if (imlist.size() % 2 != 0)
        throw std::runtime_error("graycodeword requires an even set of images\n");

    cv::cuda::Stream stream0;
    
    // Total number of graycode bits (pairs of captured graycode patterns)
    int n = imlist.size()/2;

    // Read first image to obtain the output array size
    cv::Size sz = cv::imread(imlist[0], 0).size();

    // Get output vector of arrays
    std::vector<cv::cuda::GpuMat>& gray_images = _code_word.getGpuMatVecRef();

    for (int k = 0; k < n; k++) {
        // Read graycoding pattern and its inverted counterpart
        cv::Mat im1 = cv::imread(imlist[2*k], 0);
        cv::Mat im2 = cv::imread(imlist[2*k+1], 0);
        // Generate a single gray map
        cv::Mat gray_h = (im1 > im2)/255;

        // Convert to GPU with continuous memory block of byte data
        cv::cuda::GpuMat gray;
        gray.upload(gray_h, stream0);
        gray_images.push_back(gray);
    }
}

void gray2dec(cv::InputArray _code_word, cv::OutputArray _dec) {
    // Obtain input vector of GpuMats
    std::vector<cv::cuda::GpuMat> code_word;
    _code_word.getGpuMatVector(code_word);

    // Number of graycoding arrays, rows, and columns of the images
    int n = code_word.size(), h = code_word[0].rows, w = code_word[0].cols;


    // Output array that store graycode words converted to decimal
    _dec.create(h, w, CV_32S);
    cv::cuda::GpuMat dec = _dec.getGpuMat();

    // Launching initDecimalArray to initialize the values of dec
    dim3 block(16, 16);
    dim3 grid((w + block.x - 1)/block.x, (h + block.y - 1)/block.y);
    initDecimalArray<<<grid, block>>>(code_word[0], dec, n);

    // Initializing the binary map. 
    // Where the Most Significant Bit (MSB) of the binary code = MSB gray code
    cv::cuda::GpuMat bin = code_word[0].clone();
    // Convert from gray code to decimal
    for (int i = 1; i < n; i++)
        gray2dec_array<<<grid, block>>>(code_word[i], bin, dec, n, i);
}

void decode(cv::InputArray _code_word, std::vector<float>& coor, cv::InputArray _mask) {
    cv::cuda::GpuMat dec;
    gray2dec(_code_word, dec); // int
}

} // namespace sl
