#include "hip/hip_runtime.h"
#include <SLutils/fringe_analysis.hpp>

#include <opencv2/cudaarithm.hpp>


namespace sl {

__global__ void N_phase(const cv::cuda::PtrStepSz<double> sumIcos, const cv::cuda::PtrStep<double> sumIsin,
                        cv::cuda::PtrStep<double> phase) {
    int j = blockIdx.x*blockDim.x + threadIdx.x;
    int i = blockIdx.y*blockDim.y + threadIdx.y;
    if (i >= sumIcos.rows || j >= sumIcos.cols) return;

    phase(i,j) = -atan2(sumIsin(i,j), sumIcos(i,j));
}

__global__ void N_modulation(const cv::cuda::PtrStepSz<double> sumI,
                             const cv::cuda::PtrStep<double> sumIcos,
                             const cv::cuda::PtrStep<double> sumIsin,
                             cv::cuda::PtrStep<double> data_modulation) {
    int j = blockIdx.x*blockDim.x + threadIdx.x;
    int i = blockIdx.y*blockDim.y + threadIdx.y;
    if (i >= sumI.rows || j >= sumI.cols) return;
    
    double a1 = sumIcos(i,j);
    double a2 = sumIsin(i,j);
    
    // Estimate numerator: sqrt(sumIcos^2 + sumIsin^2)
    double numerator = hypot(a1, a2); // sqrt(a1*a1 + a2*a2);
    // Final data modulation: sqrt(sumIcos^2 + sumIsin^2)/sumI
    data_modulation(i,j) = numerator/sumI(i,j);
}

__global__ void three_phase(const cv::cuda::PtrStepSzb im1, const cv::cuda::PtrStepb im2,
                            const cv::cuda::PtrStepb im3, cv::cuda::PtrStep<double> phi) {
    int j = blockIdx.x*blockDim.x + threadIdx.x;
    int i = blockIdx.y*blockDim.y + threadIdx.y;
    if (i >= im1.rows || j >= im1.cols) return;
    
    double I1 = static_cast<double>(im1(i,j));
    double I2 = static_cast<double>(im2(i,j));
    double I3 = static_cast<double>(im3(i,j));
    
    double y = sqrt(3.)*(I1 - I3);
    double x = 2*I2 - I1 - I3;
    
    phi(i,j) = atan2(y, x);
}

__global__ void three_phase_modulation(const cv::cuda::PtrStepSzb im1,
                                       const cv::cuda::PtrStepb im2,
                                       const cv::cuda::PtrStepb im3,
                                       cv::cuda::PtrStep<double> phi, cv::cuda::PtrStep<double> gamma) {
    int j = blockIdx.x*blockDim.x + threadIdx.x;
    int i = blockIdx.y*blockDim.y + threadIdx.y;
    if (i >= im1.rows || j >= im1.cols) return;
    
    double I1 = static_cast<double>(im1(i,j));
    double I2 = static_cast<double>(im2(i,j));
    double I3 = static_cast<double>(im3(i,j));
    
    double y = sqrt(3.)*(I1 - I3);
    double x = 2*I2 - I1 - I3;
    
    // Phase map
    phi(i,j) = atan2(y, x);
    
    // Data modulation
    double numerator = hypot(x, y);
    gamma(i,j) = numerator/(I1 + I2 + I3);
}


void NStepPhaseShifting(const std::vector<std::string>& imgs, cv::OutputArray _phase, int N) {
    cv::cuda::Stream stream0;

    // Initialize sumIsin and sumIcos with the first fringe image
    cv::Mat I_h = cv::imread(imgs[0], 0);
    I_h.convertTo(I_h, CV_64F);
    cv::cuda::GpuMat I(I_h);
    double delta = 2*CV_PI/N; // delta for i = 0
    
    cv::cuda::GpuMat sumIsin;
    cv::cuda::multiply(I, std::sin(delta), sumIsin, 1, -1, stream0);
    
    cv::cuda::GpuMat sumIcos;
    cv::cuda::multiply(I, std::cos(delta), sumIcos, 1, -1, stream0);
    
    
    // Add the other fringes to sumIsin and sumIcos
    for (int i = 1; i < imgs.size(); i++) {
        cv::Mat I_h = cv::imread(imgs[i], 0);
        I_h.convertTo(I_h, CV_64F);
        cv::cuda::GpuMat I(I_h);
        double delta = 2*CV_PI*(i + 1)/N;

        cv::cuda::scaleAdd(I, std::sin(delta), sumIsin, sumIsin, stream0); // sumIsin += I*std::sin(delta);
        cv::cuda::scaleAdd(I, std::cos(delta), sumIcos, sumIcos, stream0); // sumIcos += I*std::cos(delta);
    }
    
    // Set output wrapped phase array
    _phase.create(sumIsin.size(), sumIsin.type());
    cv::cuda::GpuMat phase = _phase.getGpuMat();
    
    // Estimate final wrapped phase with atan2
    dim3 block(16, 16);
    dim3 grid((phase.cols + block.x - 1)/block.x, (phase.rows + block.y - 1)/block.y);
    N_phase<<<grid, block>>>(sumIcos, sumIsin, phase);
}

void NStepPhaseShifting_modulation(const std::vector<std::string>& imgs, cv::OutputArray _phase,
                                   cv::OutputArray _data_modulation, int N) {
    cv::cuda::Stream stream0;

    // Initialize sumI, sumIsin, and sumIcos using the first fringe image
    cv::Mat sumI_h = cv::imread(imgs[0], 0);
    sumI_h.convertTo(sumI_h, CV_64F);
    cv::cuda::GpuMat sumI(sumI_h);
    double delta = 2*CV_PI/N; // delta for i = 0
    
    cv::cuda::GpuMat sumIsin;
    cv::cuda::multiply(sumI, std::sin(delta), sumIsin, 1, -1, stream0);
    
    cv::cuda::GpuMat sumIcos;
    cv::cuda::multiply(sumI, std::cos(delta), sumIcos, 1, -1, stream0);
    
    
    // Add the other fringes to sumI, sumIsin, and sumIcos
    for (int i = 1; i < imgs.size(); i++) {
        cv::Mat I_h = cv::imread(imgs[i], 0);
        I_h.convertTo(I_h, CV_64F);
        cv::cuda::GpuMat I(I_h);
        double delta = 2*CV_PI*(i + 1)/N;
        
        cv::cuda::add(sumI, I, sumI, {}, -1, stream0); // sumI += I;
        cv::cuda::scaleAdd(I, std::sin(delta), sumIsin, sumIsin, stream0); // sumIsin += I*std::sin(delta);
        cv::cuda::scaleAdd(I, std::cos(delta), sumIcos, sumIcos, stream0); // sumIcos += I*std::cos(delta);
    }
    
    // ------------- Estimate final wrapped phase with atan2
    _phase.create(sumIsin.size(), sumIsin.type());
    cv::cuda::GpuMat phase = _phase.getGpuMat();
    dim3 block(16, 16);
    dim3 grid((phase.cols + block.x - 1)/block.x, (phase.rows + block.y - 1)/block.y);
    N_phase<<<grid, block>>>(sumIcos, sumIsin, phase);
    
    
    // ----------- Estimate data modulation: sqrt(sumIcos^2 + sumIsin^2)/sumI
    cv::cuda::GpuMat numerator;
    cv::cuda::sqr(sumIcos, sumIcos, stream0); // sumIcos^2
    cv::cuda::sqr(sumIsin, sumIsin, stream0); // sumIsin^2
    cv::cuda::add(sumIcos, sumIsin, numerator, {}, -1, stream0); // sumIcos^2 + sumIsin^2
    cv::cuda::sqrt(numerator, numerator, stream0); // sqrt(sumIcos^2 + sumIsin^2)
    cv::cuda::divide(numerator, sumI, _data_modulation, 1, -1, stream0);
}

void ThreeStepPhaseShifting(const std::vector<std::string>& imgs, cv::OutputArray _phase) {
    cv::cuda::Stream stream0;
    
    // Read the three fringe images
    cv::cuda::GpuMat im1, im2, im3;
    
    cv::Mat im1_h = cv::imread(imgs[0], 0);
    im1.upload(im1_h, stream0);
    
    cv::Mat im2_h = cv::imread(imgs[1], 0);
    im2.upload(im2_h, stream0);
    
    cv::Mat im3_h = cv::imread(imgs[2], 0);
    im3.upload(im3_h, stream0);
    
    
    // Set output wrapped phase array
    _phase.create(im1.size(), CV_64F);
    cv::cuda::GpuMat phase = _phase.getGpuMat();
    
    // Estimate final wrapped phase with atan2
    dim3 block(16, 16);
    dim3 grid((phase.cols + block.x - 1)/block.x, (phase.rows + block.y - 1)/block.y);
    three_phase<<<grid, block>>>(im1, im2, im3, phase);
}

void ThreeStepPhaseShifting_modulation(const std::vector<std::string>& imgs, cv::OutputArray _phase,
                                       cv::OutputArray _data_modulation) {
    cv::cuda::Stream stream0;
    
    // Read the three fringe images
    cv::cuda::GpuMat im1, im2, im3;
    
    cv::Mat im1_h = cv::imread(imgs[0], 0);
    im1.upload(im1_h, stream0);
    
    cv::Mat im2_h = cv::imread(imgs[1], 0);
    im2.upload(im2_h, stream0);
    
    cv::Mat im3_h = cv::imread(imgs[2], 0);
    im3.upload(im3_h, stream0);
    
    
    // Set output wrapped phase array
    _phase.create(im1.size(), CV_64F);
    cv::cuda::GpuMat phase = _phase.getGpuMat();
    
    // Set output data modulation array
    _data_modulation.create(phase.size(), phase.type());
    cv::cuda::GpuMat data_modulation = _data_modulation.getGpuMat();
    
    // Estimate final wrapped phase and data modulation arrays
    dim3 block(16, 16);
    dim3 grid((phase.cols + block.x - 1)/block.x, (phase.rows + block.y - 1)/block.y);
    three_phase_modulation<<<grid, block>>>(im1, im2, im3, phase, data_modulation);
}

} // namespace sl
